#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""
#include "hip/device_functions.h"
#include "..\MayaBoids\RulesSetup.h"

#include "GPU_DataStructuresOperations.cuh"	//inlined function
#include "GPU_Decision.cuh"
#include "GPU_Desires.cuh"
#include "GPU_Locomotion.cuh"

extern "C" void GPUCompute(Boid *boids);
extern "C" int GPUInit(SimulationParameters simParam, RulesParameters *rulesP, int nDes);

/* device global variables */
__device__ __constant__ Vector arrivalPoint;	
__device__ __constant__ RulesParameters rP[MAXNUMBEROFDESIRES];


// host global variables
int nBoids;				// number of boids
uint3 dimBlock;			// cuda resources simulation: block dimension
uint3 dimGrid;			// cuda resources simulation: grid dimension

/*
*********************************************
*     Flock simulation					  *
*********************************************
*/

// locomotion function
__device__  void locomotion(Vector *decisionVector,Boid *currentBoid, double dt)
{
	cudaAddVector(&currentBoid->currentVelocity,decisionVector,&currentBoid->nextVelocity);
	cudaLimitationVector(&currentBoid->nextVelocity, currentBoid->maxSpeed);
	cudaAddVector(&currentBoid->currentPosition,&currentBoid->nextVelocity,&currentBoid->nextPosition);
}

// steering function
__device__ void steering(Boid * currentBoid, Vector *target, Vector *steer)
{
	Vector desired;
	double desiredMagnitude;

	cudaInitVector(steer);
	cudaSubVector(target,&(currentBoid->currentPosition),&desired);
	desiredMagnitude=cudaMagnitudeVector(&desired);

	if(desiredMagnitude>0)
	{
		cudaNormalizeVector(&desired);
		//max speed option
		cudaMultiplyVector(&desired,currentBoid->maxSpeed,&desired);
		cudaSubVector(&desired,&(currentBoid->currentVelocity) ,steer);
		cudaLimitationVector(steer,currentBoid->maxForce);
	}	
}

// COHESION RULE IMPLEMENTATION
__device__ void cohesionRule(Boid*cudaBoidSet, Boid * currentBoid, Vector * steer, int nBoids)
{
	Vector tempVector;
	int count;

	cudaInitVector(&tempVector);
	cudaInitVector(steer);
	count=0;

	for (int i=0;i<nBoids;i++)
	{
		Boid *thisBoid = &cudaBoidSet[i];
		if((thisBoid->id!=currentBoid->id) && (cudaDistanceVector(&thisBoid->currentPosition,&currentBoid->currentPosition)<rP[COHESIONRULE].ruleRadius))
		{
			cudaAddVector(&(thisBoid->currentPosition), &tempVector, &tempVector);
			count++;
		}
	}
	if(count>0)
	{
		cudaMultiplyVector(&tempVector,1.0f/count,&tempVector);
		steering(currentBoid, &tempVector, steer);
		cudaMultiplyVector(steer, rP[COHESIONRULE].ruleFactor,steer);
	}
}

// SEPARATION RULE IMPLEMENTATION
__device__ void separationRule(Boid*cudaBoidSet, Boid * currentBoid, Vector * steer, int nBoids)
{
	Vector tempVector;
	int count;

	cudaInitVector(steer);
	cudaInitVector(&tempVector);
	count=0;

	for (int i=0;i<nBoids;i++)
	{
		double dis=0;
		Boid *thisBoid = &cudaBoidSet[i];
		if((thisBoid->id!=currentBoid->id) && (cudaDistanceVector(&thisBoid->currentPosition,&currentBoid->currentPosition)<rP[SEPARATIONRULE].ruleRadius))
		{
			cudaSubVector(&(thisBoid->currentPosition),&(currentBoid->currentPosition),&tempVector);
			dis = cudaMagnitudeVector(&tempVector);
			cudaMultiplyVector(&tempVector,1.0f/dis,&tempVector);
			cudaSubVector(steer,&tempVector,steer);
			count++;
		}
	}
	if(count>0)
	{
		cudaMultiplyVector(steer,1.0f/count,steer);
		if(cudaMagnitudeVector(steer)>0)
		{	
			cudaNormalizeVector(steer);
			cudaMultiplyVector(steer,currentBoid->maxSpeed,steer);
			cudaSubVector(steer,&(currentBoid->currentVelocity),steer);
			cudaLimitationVector(steer,currentBoid->maxForce);
			cudaMultiplyVector(steer, rP[SEPARATIONRULE].ruleFactor,steer);
		}
	}
}

// ALIGNMENT RULE IMPLEMENTATION
__device__ void alignmentRule(Boid*cudaBoidSet, Boid * currentBoid, Vector * steer, int nBoids)
{
	int count=0;
	cudaInitVector(steer);

	for(int i=0;i<nBoids;i++)
	{
		Boid *thisBoid = &cudaBoidSet[i];
		if((thisBoid->id!=currentBoid->id) && (cudaDistanceVector(&thisBoid->currentPosition,&currentBoid->currentPosition)<rP[ALIGNMENTRULE].ruleRadius))
		{	
			cudaAddVector(&(thisBoid->currentVelocity), steer, steer);
			count++;
		}
	}

	if(count>0)
	{
		cudaMultiplyVector(steer,1.0f/count,steer);
		if(cudaMagnitudeVector(steer)>0)
		{
			cudaNormalizeVector(steer);
			cudaMultiplyVector(steer,currentBoid->maxSpeed,steer);
			cudaSubVector(steer,&(currentBoid->currentVelocity),steer);
			cudaLimitationVector(steer,currentBoid->maxForce);
			cudaMultiplyVector(steer, rP[ALIGNMENTRULE].ruleFactor,steer);
		}
	}
}

// Reach RULE IMPLEMENTATION
__device__ void reachRule(Boid *currentBoid, Vector *outputVector)
{
	cudaInitVector(outputVector);
	steering(currentBoid, &arrivalPoint, outputVector);
	cudaMultiplyVector(outputVector, rP[REACHRULE].ruleFactor,outputVector);
}

// Follow RULE IMPLEMENTATION
__device__ void followRule(Boid *currentBoid,Boid* leader, Vector *outputVector)
{
	cudaInitVector(outputVector);
	if(leader!=NULL)
	{
		steering(currentBoid, &leader->currentPosition, outputVector);
		cudaMultiplyVector(outputVector, rP[FOLLOWRULE].ruleFactor,outputVector);
	}
}
// boids update method
__global__ void GPUupdateBoids(Boid* cboids, int nb)
{
	Vector temp,tot;
	cudaInitVector(&temp);
	cudaInitVector(&tot);

	// brute force implementation
	int tx =  blockIdx.x * blockDim.x + threadIdx.x;
	
	cohesionRule(cboids,&cboids[tx],&tot,nb);
	cudaMultiplyVector(&tot, rP[COHESIONRULE].ruleWeight,&tot);

	alignmentRule(cboids,&cboids[tx],&temp,nb);
	cudaMultiplyVector(&temp, rP[ALIGNMENTRULE].ruleWeight,&temp);
	cudaAddVector(&temp,&tot,&tot);

	separationRule(cboids,&cboids[tx],&temp,nb);
	cudaMultiplyVector(&temp, rP[SEPARATIONRULE].ruleWeight ,&temp);
	cudaAddVector(&temp,&tot,&tot);

	locomotion(&tot,&cboids[tx],0.1);

	__syncthreads();

	cudaCopyVector(&cboids[tx].nextPosition,&cboids[tx].currentPosition);
	cudaCopyVector(&cboids[tx].nextVelocity,&cboids[tx].currentVelocity);
}


// initializing variables
int GPUInit(SimulationParameters simParam, RulesParameters *rulesP, int nDes)
{
	// hardware capability and resources
	int deviceCount,nBlocks;
	hipDeviceProp_t deviceCuda;
	
	hipGetDeviceCount(&deviceCount);
	if (deviceCount>0)
		for(int i=0;i<deviceCount;i++)
		{
			hipGetDeviceProperties(&deviceCuda,i);
			if(deviceCuda.major>=2)
			{
				if (simParam.numberOfBoids>deviceCuda.maxThreadsPerBlock)
				{
					// managing the right procedure for computing boids 
 					nBlocks =(simParam.numberOfBoids +deviceCuda.maxThreadsPerBlock-1)/deviceCuda.maxThreadsPerBlock ;
			
					dimBlock.x=(simParam.numberOfBoids/nBlocks);
					dimBlock.y=1;
					dimBlock.z=1;

					dimGrid.x=nBlocks;
					dimGrid.y=1;
					dimGrid.z=1;
				}
				else
				{
					dimBlock.x=simParam.numberOfBoids;
					dimBlock.y=1;
					dimBlock.z=1;

					dimGrid.x=1;
					dimGrid.y=1;
					dimGrid.z=1;
				}

				// saving host variable
				nBoids=simParam.numberOfBoids;
				// passing viariables to device 
				hipMemcpyToSymbol(HIP_SYMBOL(rP),rulesP,nDes*sizeof(RulesParameters));
			}
			return CUDA_DEVICE_NOT_SUPPORTED;
		}
	else
		return NO_CUDA_DEVICE;

	return OKCUDA;
}

// computing values on cuda device
void GPUCompute(Boid *boids)
{
	Boid * d_boids;

	dim3 threadB(dimBlock);
	dim3 gridB(dimGrid);
	
	hipMalloc((void**)&d_boids,nBoids*sizeof(Boid));
	hipMemcpy(d_boids,boids,nBoids*sizeof(Boid),hipMemcpyHostToDevice);
	GPUupdateBoids<<<gridB,threadB>>>(d_boids,nBoids);			//gpu computing
	//retrive data (copy data from device memory to host memory)
	hipMemcpy(boids,d_boids,nBoids*sizeof(Boid),hipMemcpyDeviceToHost);
	//free device resources
	hipFree(d_boids);
}
