#include "hip/hip_runtime.h"
#include "GPU_DataStructuresOperations.cuh"
/**/
/*
*********************************************
*     "Vector" struct operation            *
*********************************************
*/

//__device__ void initVector(Vector *v)
//{
//	v->x=0;v->y=0;v->z=0;
//}
//
//__device__ void copyVector(const Vector *inputVector, Vector *outputVector)
//{
//	outputVector->x=inputVector->x;
//	outputVector->y=inputVector->y;
//	outputVector->z=inputVector->z;
//}
//
//__device__ void addVector(Vector *v1,Vector *v2,Vector *v1plusv2)
//{
//	v1plusv2->x=v1->x + v2->x;
//	v1plusv2->y=v1->y + v2->y;
//	v1plusv2->z=v1->z + v2->z;
//}
//
//__device__ void subVector(Vector *v1,Vector *v2,Vector *v1minusv2)
//{
//	v1minusv2->x=v1->x - v2->x;
//	v1minusv2->y=v1->y - v2->y;
//	v1minusv2->z=v1->z - v2->z;
//}
//
//__device__ void multiply(Vector *vettore, double m,Vector *outputVector)
//{
//	outputVector->x=m*vettore->x;
//	outputVector->y=m*vettore->y;
//	outputVector->z=m*vettore->z;
//}
//
//__device__ double magnitude(const Vector *v)
//{
//	double value = v->x * v->x + v->y * v->y + v->z * v->z;
//	return sqrt(value);
//}
//
//__device__ void normalize(Vector *v)
//{
//	double mag = magnitude(v);
//	if (mag==0)
//		return;
//	multiply(v, 1/mag, v);
//}
//
//__device__  double dotProduct(const Vector *v1,const Vector *v2){return (v1->x * v2->x + v1->y * v2->y + v1->z * v2->z ); }
//
//__device__  double angleBetween(const Vector *v1, const Vector *v2)
//{
//
//	double p1 = dotProduct(v1, v2);
//	double p2 = magnitude(v1) * magnitude(v2);
//	double p3 = acos(p1/p2);
//	return p3;
//}
//
//__device__ BOOL isNotEqual(const Vector *v1,const Vector *v2){ return(v1->x != v2->x || v1->y != v2->y || v1->z != v2->z); }
//
//__device__ BOOL isEqual(const Vector *v1,const Vector *v2){ return(v1->x == v2->x && v1->y == v2->y && v1->z == v2->z); }
//
//__device__ void crossProduct(const Vector *v1,const Vector *v2, Vector *outputVector)
//{
//	outputVector->x = v1->y * v2->z - v1->z * v2->y;
//	outputVector->y = v1->z * v2->x - v1->x * v2->z;
//	outputVector->z = v1->x * v2->y - v1->y * v2->x;
//}
//
//__device__ void direction(const Vector *v, Vector *oVector)
//{
//	copyVector(v, oVector);
//	normalize(oVector);
//}
//
//__device__ void setMagnitude(Vector *v, double magnitude)
//{
//	normalize(v);
//	multiply(v, magnitude, v);
//}
//
//__device__ void copyMagnitude(const Vector *inputVector, Vector *outputVector)
//{
//	double mag = magnitude(inputVector);
//	setMagnitude(outputVector, mag);
//}
//
//__device__ void setDirection(Vector *v)
//{
//	double m = magnitude(v);
//	Vector *d = v;
//	normalize(d);
//	multiply(d, m, d);
//	copyVector(d, v);
//}
//
//__device__ void copyDirection(const Vector *inputVector, Vector *outputVector)
//{
//	double m = magnitude(outputVector);
//	Vector copyV;
//	copyVector(inputVector, &copyV);
//	normalize(&copyV);
//	multiply(&copyV, m, outputVector);
//}
//
//__device__ void limitation(Vector *inputVector, double limit)
//{
//	if(magnitude(inputVector)>limit)
//	{
//		normalize(inputVector);
//		multiply(inputVector,limit,inputVector);
//	}
//}
//
//__device__ double distance(Vector *v1, Vector*v2)
//{
//	Vector vd;
//	subVector(v1,v2,&vd);
//	return magnitude(&vd);
//}
//
////*
////*********************************************
////*     "Boid" struct operation					*
////*********************************************
////*/
//
//__device__ void copyBoid(const Boid *boidSource,Boid *boidDestination)
//{
//	copyVector(&(boidSource->acceleration),&(boidDestination->acceleration));
//
//	copyVector(&(boidSource->currentPosition),&(boidDestination->currentPosition));
//	copyVector(&(boidSource->nextPosition),&(boidDestination->nextPosition));
//
//	copyVector(&(boidSource->currentVelocity),&(boidDestination->currentVelocity));
//	copyVector(&(boidSource->nextVelocity),&(boidDestination->nextVelocity));
//
//	boidDestination->id=boidSource->id;
//	boidDestination->boundingBoxRadius=boidSource->boundingBoxRadius;
//	boidDestination->mass=boidSource->mass;
//	boidDestination->maxAcceleration=boidSource->maxAcceleration;
//	boidDestination->maxForce=boidSource->maxForce;
//	boidDestination->maxSpeed=boidSource->maxSpeed;
//	boidDestination->wanderTheta=boidSource->wanderTheta;
//
//
//}
//
//__device__ void initBoid(const Vector *p, const Vector *v, const Vector *a, double speed, double acceleration, double mForce, double bMass, long int id, Boid *boid)
//{
//	copyVector(p,&(boid->currentPosition));
//	copyVector(p,&(boid->nextPosition));
//	copyVector(v,&(boid->currentVelocity));
//	copyVector(p,&(boid->nextVelocity));
//	copyVector(a,&(boid->acceleration));
//
//	boid->id=id;
//	boid->mass=bMass;
//	boid->maxAcceleration=acceleration;
//	boid->maxForce=mForce;
//	boid->maxSpeed=speed;
//	boid->wanderTheta=0;
//}

